#include "hip/hip_runtime.h"
#include "init.h"
#include "common.h"

#include <string.h>
#include <stdio.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define MASK 1

/**
 * @brief Computes the number of 1's
 *
 * @param i The i-th variable that composes the hypervector
 * @return  Number of 1's in i-th variable of hypervector
 */
__device__ static inline int
number_of_set_bits(uint32_t i) {
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}

/**
 * @brief Computes the maximum Hamming Distance.
 *
 * @param[in] distances Distances associated to each class
 * @return              The class related to the maximum distance
 */
__device__ static int
max_dist_hamm(int distances[CLASSES]) {
    int max = distances[0];
    int max_index = 0;

    for (int i = 1; i < CLASSES; i++) {
        if (max > distances[i]) {
            max = distances[i];
            max_index = i;
        }
    }

    return max_index;
}

/**
 * @brief Computes the Hamming Distance for each class.
 *
 * @param[in] q     Query hypervector
 * @param[in] aM    Associative Memory matrix
 * @param[out] sims Distances' vector
 */
__device__ static void
hamming_dist(uint32_t *q, uint32_t *aM, int sims[CLASSES], gpu_hdc_vars *hd) {
    for (int i = 0; i < CLASSES; i++) {
        sims[i] = 0;
        for (int j = 0; j < hd->bit_dim + 1; j++) {
            sims[i] += number_of_set_bits(q[j] ^ aM[A2D1D(hd->bit_dim + 1, i, j)]);
        }
    }
}

/**
 * @brief Tests the accuracy based on input testing queries.
 *
 * @param[in] q_32  Query hypervector
 * @param[in] aM_32 Trained associative memory
 * @return          Classification result
 */
__device__ static int
associative_memory_32bit(uint32_t *q_32, uint32_t *aM_32, gpu_hdc_vars *hd) {
    int sims[CLASSES] = {0};

    // Computes Hamming Distances
    hamming_dist(q_32, aM_32, sims, hd);

    // Classification with Hamming Metric
    return max_dist_hamm(sims);
}

/**
 * @brief Computes the N-gram.
 *
 * @param[in] input       Input data
 * @param[out] query      Query hypervector
 */
__device__ static void
compute_N_gram(int32_t *input, uint32_t *query, gpu_hdc_vars *hd) {

    uint32_t chHV[MAX_CHANNELS + 1];

    for (int i = 0; i < hd->bit_dim + 1; i++) {
        query[i] = 0;
        for (int j = 0; j < hd->channels; j++) {
            int ix = input[j];

            uint32_t im = hd->iM[A2D1D(hd->bit_dim + 1, ix, i)];
            uint32_t cham = hd->chAM[A2D1D(hd->bit_dim + 1, j, i)];

            chHV[j] = im ^ cham;
        }
        // this is done to make the dimension of the matrix for the componentwise majority odd.
        chHV[hd->channels] = chHV[0] ^ chHV[1];

        // componentwise majority: compute the number of 1's
        for (int z = 31; z >= 0; z--) {
            uint32_t cnt = 0;
            for (int j = 0; j < hd->channels + 1; j++) {
                uint32_t a = chHV[j] >> z;
                uint32_t mask = a & 1;
                cnt += mask;
            }

            if (cnt > 2) {
                query[i] = query[i] | (1 << z);
            }
        }
    }
}


/**
 * @brief Run HDC algorithm
 * @param[out] result         Buffer to place results in
 * @param[out] result_offset  Offset to start placing results from
 * @param[in] task_begin      Position to start task from
 * @param[in] task_end        Position to end task at
 * @param[in] hd              HDC vars
 *
 * @return                    Non-zero on failure.
 */
__global__ void
gpu_hdc(gpu_input_data *gpu_data, int32_t *read_buf, int32_t *result, gpu_hdc_vars *hd) {
    uint32_t overflow = 0;
    uint32_t old_overflow = 0;

    uint32_t q[MAX_BIT_DIM + 1] = {0};
    uint32_t q_N[MAX_BIT_DIM + 1] = {0};
    int32_t quantized_buffer[MAX_CHANNELS] = {0};

    int result_num = 0;

    int thr = (blockIdx.x * blockDim.x) + threadIdx.x;

    if ((gpu_data->task_end[thr] - gpu_data->task_begin[thr]) <= 0) {
        dbg_printf("%u: No work to do\n", thr);
        return;
    }

    for (int ix = gpu_data->task_begin[thr]; ix < gpu_data->task_end[thr]; ix += hd->n) {

        for (int z = 0; z < hd->n; z++) {

            for (int j = 0; j < hd->channels; j++) {
                if (ix + z < gpu_data->buffer_channel_usable_length) {
                    int ind = A2D1D(gpu_data->buffer_channel_usable_length, j, ix + z);
                    quantized_buffer[j] = read_buf[ind];
                }
            }

            // Spatial and Temporal Encoder: computes the N-gram.
            // N.B. if N = 1 we don't have the Temporal Encoder but only the Spatial Encoder.
            if (z == 0) {
                compute_N_gram(quantized_buffer, q, hd);
            } else {
                compute_N_gram(quantized_buffer, q_N, hd);

                // Here the hypervector q is shifted by 1 position as permutation,
                // before performing the componentwise XOR operation with the new query (q_N).
                int32_t shifted_q;
                overflow = q[0] & MASK;
                for (int i = 1; i < hd->bit_dim; i++) {
                    old_overflow = overflow;
                    overflow = q[i] & MASK;
                    shifted_q = (q[i] >> 1) | (old_overflow << (32 - 1));
                    q[i] = q_N[i] ^ shifted_q;
                }

                old_overflow = overflow;
                overflow = (q[hd->bit_dim] >> 16) & MASK;
                shifted_q = (q[hd->bit_dim] >> 1) | (old_overflow << (32 - 1));
                q[hd->bit_dim] = q_N[hd->bit_dim] ^ shifted_q;

                shifted_q = (q[0] >> 1) | (overflow << (32 - 1));
                q[0] = q_N[0] ^ shifted_q;
            }
        }

        // Classifies the new N-gram through the Associative Memory matrix.
        result[gpu_data->idx_offset[thr] + result_num] = associative_memory_32bit(q, hd->aM_32, hd);
        // printf("i=%i,r=%i\n", result_num, result[gpu_data->idx_offset[thr] + result_num]);
        result_num++;
    }
}
