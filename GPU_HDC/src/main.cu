#include "aux_functions.h"
#include "common.h"
#include "host.h"
#include "host_only.h"
#include "init.h"

#include <string.h>
#include <stdbool.h>
#include <getopt.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define TIME_NOW(_t) (clock_gettime(CLOCK_MONOTONIC, (_t)))

/**
 * @struct gpu_runtime
 * @brief GPU execution times
 */
typedef struct gpu_runtime {
    double execution_time_alloc;
    double execution_time_copy_in;
    double execution_time_launch;
    double execution_time_copy_out;
} gpu_runtime;

/**
 * @struct in_buffer
 *
 * @brief   Input buffer for a DPU
 */
typedef struct in_buffer {
    int32_t buffer[HDC_MAX_INPUT];
    size_t buffer_size;
} in_buffer;

/**
 * @struct hdc_data
 * @brief HDC data for HDC task
 */
typedef struct hdc_data {
    int32_t *data_set;     /**< Input HDC dataset */
    int32_t *results;      /**< Output from run */
    uint32_t result_len;   /**< Length of the results */
    double execution_time; /**< Total execution time of run */
} hdc_data;

/**
 * @brief Function for @p run_hdc to run HDC task
 */
typedef int (*hdc)(hdc_data *data, void *runtime);

#define gpuErrchk(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}

/**
 * @brief Run a HDC workload and time the execution
 *
 * @param[in] fn        Function to run HDC algorithm
 * @param[out] data     Results from HDC run
 * @param[out] runtime  Run times from sections of @p fn
 *
 * @return Non-zero On failure
 */
static double
run_hdc(hdc fn, hdc_data *data, void *runtime) {
    struct timespec start, end;

    int ret = 0;

    uint8_t extra_result = (number_of_input_samples % hd.n) != 0;
    data->result_len = (number_of_input_samples / hd.n) + extra_result;
    uint32_t result_size = data->result_len * sizeof(int32_t);

    if ((data->results = (int32_t *)malloc(result_size)) == NULL) {
        nomem();
    }

    TIME_NOW(&start);
    ret = fn(data, runtime);
    TIME_NOW(&end);

    data->execution_time = TIME_DIFFERENCE(start, end);

    return ret;
}

/**
 * @brief Compare the results from the host and DPU confirming they are the same
 *        or printing differences
 *
 * @param[in] dpu_data   Results to be tested from DPU
 * @param[in] host_data  Results to be tested from host
 * @param[in] check_only Only check results are equal, dont print differences
 *
 * @return               Non-zero if results are not the same
 */
static int
compare_results(hdc_data *dpu_data, hdc_data *host_data, bool check_only) {
    int ret = 0;

    if (!check_only) {
        printf("--- Compare --\n");
        printf("(%u) results\n", host_data->result_len);
    }

    for (uint32_t i = 0; i < host_data->result_len; i++) {
        if (host_data->results[i] != dpu_data->results[i]) {
            if (check_only) {
                return -1;
            }
            fprintf(stderr, "(host_results[%u] = %d) != (dpu_results[%u] = %d)\n", i,
                    host_data->results[i], i, dpu_data->results[i]);
            ret = -1;
        }
    }

    if (check_only) {
        return 0;
    }

    char *faster;
    double time_diff, percent_diff;
    if (dpu_data->execution_time > host_data->execution_time) {
        faster = "Host";
        time_diff = dpu_data->execution_time - host_data->execution_time;
        percent_diff = dpu_data->execution_time / host_data->execution_time;
    } else {
        faster = "GPU";
        time_diff = host_data->execution_time - dpu_data->execution_time;
        percent_diff = host_data->execution_time / dpu_data->execution_time;
    }

    printf("%s was %fs (%f x) faster\n", faster, time_diff, percent_diff);

    return ret;
}

/**
 * @brief Print results from HDC run
 * @param[in] data  Results to print
 */
static void
print_results(hdc_data *data) {
    for (uint32_t i = 0; i < data->result_len; i++) {
        printf("%d\n", data->results[i]);
    }
}

/**
 * @brief Calculate individual buffer lengths for each DPU or tasklet with as even distribution as
 * possible
 * @param[in]  length                  Lengths of @p buffer_channel_lengths
 * @param[in]  samples                 Samples to distribute
 * @param[out] buffer_channel_lengths  Lengths for each channel
 */
static void
calculate_buffer_lengths(uint32_t length, uint32_t *buffer_channel_lengths,
                         uint32_t input_samples) {
    /* Section of buffer for one channel, without samples not divisible by n */
    uint32_t samples = input_samples / length;
    /* Remove samples not divisible by n */
    uint32_t chunk_size = samples - (samples % hd.n);
    /* Extra data for last DPU */
    uint32_t extra_data = input_samples - (chunk_size * length);
    uint32_t extra_data_divisible = extra_data / hd.n;

    for (uint32_t i = 0; i < length; i++) {
        buffer_channel_lengths[i] = chunk_size;
    }

    uint32_t i = 0;
    while (extra_data_divisible != 0) {
        buffer_channel_lengths[i] += hd.n;
        extra_data_divisible--;
        i++;
        if (i == length) {
            i = 0;
        }
    }
    buffer_channel_lengths[length - 1] += extra_data % hd.n;
}

/**
 * @brief Set up the data for each GPU block
 * @param[out]    input                  Datastructure to be populated for GPU
 * @param[in]     buffer_channel_length  Length of an individual channel
 * @param[in]     data_in                Data buffer for DPU
 * @param[in]     data_set               Input data for populating @p data_in
 * @param[in,out] buff_offset            Current offset in @p dataset
 * @param[in]     gpu_id                 ID of DPU
 *
 * @return                               Non-zero on failure
 */
static int
setup_gpu_data(gpu_input_data *input, uint32_t buffer_channel_length,
               int32_t *data_set, uint32_t *buff_offset, uint32_t gpu_id) {

    uint32_t num_splits = NR_BLOCKS*NR_THREADS;

    uint32_t buffer_channel_lengths[NR_BLOCKS*NR_THREADS];
    calculate_buffer_lengths(NR_BLOCKS*NR_THREADS, buffer_channel_lengths, buffer_channel_length);

    uint32_t loc = 0;
    uint32_t idx_offset = 0;
    for (uint32_t idx = 0; idx < num_splits; idx++) {
        input->task_begin[idx] = loc;
        loc += buffer_channel_lengths[idx];
        input->task_end[idx] = loc;

        uint32_t task_samples = input->task_end[idx] - input->task_begin[idx];

        input->idx_offset[idx] = idx_offset;
        idx_offset += task_samples / hd.n;

        dbg_printf("%u: samples = %u\n", idx, task_samples);
        dbg_printf("%u: idx_offset = %u\n", idx, input->idx_offset[idx]);
        dbg_printf("%u: task_end = %u, task_begin = %u\n", idx, input->task_end[idx],
                   input->task_begin[idx]);
    }

    /* Input */
    if (gpu_id == NR_BLOCKS - 1) {
        /* No n on last in algorithm */
        input->buffer_channel_usable_length = buffer_channel_length;
    } else {
        input->buffer_channel_usable_length = buffer_channel_length + hd.n;
    }
    input->buffer_channel_aligned_size = ALIGN(buffer_channel_length * sizeof(int32_t), 8);

    /* Output */
    uint32_t extra_result = (buffer_channel_length % hd.n) != 0;
    input->output_buffer_length = (buffer_channel_length / hd.n) + extra_result;

    input->buffer_channel_length = buffer_channel_length;

    *buff_offset += input->buffer_channel_length;

    size_t total_xfer = 0;
    if (total_xfer > (HDC_MAX_INPUT * sizeof(int32_t))) {
        fprintf(stderr, "Error %lu is too large for read_buf[%d]\n", total_xfer / sizeof(int32_t),
                HDC_MAX_INPUT);
        return -1;
    }

    return 0;
}

/**
 * @brief Run the HDC algorithm for the GPU
 *
 * @param[in]  data_set  Input dataset
 * @param[out] results   Results from run
 * @param[out] runtime   Runtimes of individual sections (unused)
 *
 * @return               Non-zero on failure.
 */
static int
gpu_setup_hdc(hdc_data *data, void *runtime) {

    uint32_t buff_offset = 0;

    gpu_runtime *rt = (gpu_runtime *)runtime;

    struct timespec start, end;

    gpu_input_data *g_inputs;
    gpu_hdc_vars *g_hd;
    int32_t *g_results;

    TIME_NOW(&start);
    uint32_t result_size = data->result_len * sizeof(int32_t);
    gpuErrchk(hipMalloc((void **)&g_results, result_size));
    gpuErrchk(hipMallocManaged((void **)&g_inputs, sizeof(gpu_input_data), hipMemAttachGlobal));
    gpuErrchk(hipMallocManaged((void **)&g_hd, sizeof(gpu_hdc_vars), hipMemAttachGlobal))

    hipDeviceSynchronize();
    TIME_NOW(&end);

    rt->execution_time_alloc = TIME_DIFFERENCE(start, end);

    TIME_NOW(&start);
    memcpy(g_hd, &hd, sizeof(gpu_hdc_vars));
    memcpy(g_hd->iM, iM, MAX_IM_LENGTH * (MAX_BIT_DIM + 1) * sizeof(uint32_t));
    memcpy(g_hd->chAM, chAM, MAX_CHANNELS * (MAX_BIT_DIM + 1) * sizeof(uint32_t));

    // Copy in:
    setup_gpu_data(g_inputs, number_of_input_samples,
                   data->data_set, &buff_offset, NR_BLOCKS-1);
    TIME_NOW(&end);

    rt->execution_time_copy_in = TIME_DIFFERENCE(start, end);

    TIME_NOW(&start);
    gpu_hdc<<<NR_BLOCKS,NR_THREADS>>>(g_inputs, data->data_set, g_results, g_hd);

    hipDeviceSynchronize();
    TIME_NOW(&end);

    rt->execution_time_launch = TIME_DIFFERENCE(start, end);

    TIME_NOW(&start);
    gpuErrchk(hipMemcpy((void *)data->results, (void *)g_results, result_size, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    TIME_NOW(&end);


    rt->execution_time_copy_out = TIME_DIFFERENCE(start, end);

    gpuErrchk(hipFree(g_inputs));
    gpuErrchk(hipFree(g_hd));

    return 0;
}

/**
 * @brief Run the HDC algorithm for the host
 *
 * @param[in]  data_set  Input dataset
 * @param[out] results   Results from run
 * @param[out] runtime   Runtimes of individual sections (unused)
 *
 * @return               Non-zero on failure.
 */
static int
host_hdc(hdc_data *data, void *runtime) {

    (void) runtime;

    uint32_t overflow = 0;
    uint32_t old_overflow = 0;
    uint32_t mask = 1;
    uint32_t q[hd.bit_dim + 1];
    uint32_t q_N[hd.bit_dim + 1];
    int32_t quantized_buffer[hd.channels];

    int result_num = 0;

    for (int ix = 0; ix < number_of_input_samples; ix += hd.n) {

        for (int z = 0; z < hd.n; z++) {

            for (int j = 0; j < hd.channels; j++) {
                if (ix + z < number_of_input_samples) {
                    int ind = A2D1D(number_of_input_samples, j, ix + z);
                    quantized_buffer[j] = data->data_set[ind];
                }
            }

            // Spatial and Temporal Encoder: computes the n-gram.
            // N.B. if n = 1 we don't have the Temporal Encoder but only the Spatial Encoder.
            if (z == 0) {
                host_compute_N_gram(quantized_buffer, q);
            } else {
                host_compute_N_gram(quantized_buffer, q_N);

                // Here the hypervector q is shifted by 1 position as permutation,
                // before performing the componentwise XOR operation with the new query (q_N).
                overflow = q[0] & mask;

                for (int i = 1; i < hd.bit_dim; i++) {
                    old_overflow = overflow;
                    overflow = q[i] & mask;
                    q[i] = (q[i] >> 1) | (old_overflow << (32 - 1));
                    q[i] = q_N[i] ^ q[i];
                }

                old_overflow = overflow;
                overflow = (q[hd.bit_dim] >> 16) & mask;
                q[hd.bit_dim] = (q[hd.bit_dim] >> 1) | (old_overflow << (32 - 1));
                q[hd.bit_dim] = q_N[hd.bit_dim] ^ q[hd.bit_dim];

                q[0] = (q[0] >> 1) | (overflow << (32 - 1));
                q[0] = q_N[0] ^ q[0];
            }
        }
        // classifies the new N-gram through the Associative Memory matrix.
        data->results[result_num] = host_associative_memory_32bit(q, hd.aM_32);
        // printf("i=%i,r=%i\n", result_num, results[result_num]);
        result_num++;
    }

    return 0;
}

/**
 * @brief Display usage information to @p stream
 * @param[in] stream    File pointer to write usage to
 * @param[in] exe_name  Name of executable
 */
static void
usage(FILE *stream, char const *exe_name) {
#ifdef DEBUG
    fprintf(stream, "**DEBUG BUILD**\n");
#endif

    fprintf(stream, "usage: %s [ -d ] -i <INPUT_FILE>\n", exe_name);
    fprintf(stream, "\ti: input file\n");
    fprintf(stream, "\tr: show runtime only\n");
    fprintf(stream, "\ts: show results\n");
    fprintf(stream, "\tt: test results\n");
    fprintf(stream, "\th: help message\n");
}

int
main(int argc, char **argv) {
    bool use_gpu = false;
    bool show_results = false;
    bool test_results = false;
    bool runtime_only = false;
    int ret = 0;
    int host_ret = 0;
    int gpu_ret = 0;
    char const options[] = "sgthri:";
    char *input = NULL;

    int opt;
    while ((opt = getopt(argc, argv, options)) != -1) {
        switch (opt) {
            case 'i':
                input = optarg;
                break;

            case 'g':
                use_gpu = true;
                break;

            case 's':
                show_results = true;
                break;

            case 't':
                test_results = true;
                break;

            case 'r':
                runtime_only = true;
                break;

            case 'h':
                usage(stdout, argv[0]);
                return EXIT_SUCCESS;

            default:
                usage(stderr, argv[0]);
                return EXIT_FAILURE;
        }
    }

    if (input == NULL) {
        fprintf(stderr, "Please add an input file\n");
        usage(stderr, argv[0]);
        return EXIT_FAILURE;
    }

    double *test_set;
    ret = read_data(input, &test_set);
    if (ret != 0) {
        return ret;
    }

    uint32_t buffer_size = (sizeof(int32_t) * number_of_input_samples * hd.channels);
    int32_t *data_set = (int32_t *)malloc(buffer_size);
    if (data_set == NULL) {
        nomem();
    }

    int32_t *g_data_set;
    gpuErrchk(hipMallocManaged((void **)&g_data_set, buffer_size, hipMemAttachGlobal));

    quantize_set(test_set, data_set);

    memcpy(g_data_set, data_set, buffer_size);

    hdc_data gpu_results = {.data_set = g_data_set, .results = NULL};
    hdc_data host_results = {.data_set = data_set, .results = NULL};

    gpu_runtime rt;

    if (test_results) {
        host_ret = run_hdc(host_hdc, &host_results, NULL);
        if (host_ret != 0) {
            goto err;
        }
    }

    if (use_gpu) {
        gpu_ret = run_hdc(gpu_setup_hdc, &gpu_results, &rt);
        if (gpu_ret != 0) {
            goto err;
        }
    }

    if (test_results) {
        if (!runtime_only) {
            printf("--- Host --\n");
            if (show_results) {
                print_results(&host_results);
            }
            printf("Host took %fs\n", host_results.execution_time);
        } else {
            printf("%f\n", host_results.execution_time);
        }
    }

    if (use_gpu || test_results) {
        if (!runtime_only) {
            printf("--- GPU --\n");
            if (show_results) {
                print_results(&gpu_results);
            }
            printf("GPU took %fs\n", gpu_results.execution_time);
            printf("GPU alloc took %fs\n", rt.execution_time_alloc);
            printf("GPU copy_in took %fs\n", rt.execution_time_copy_in);
            printf("GPU launch took %fs\n", rt.execution_time_launch);
            printf("GPU copy_out took %fs\n", rt.execution_time_copy_out);
        } else {
            printf("%f,%f,%f,%f,%f\n", gpu_results.execution_time,
                   rt.execution_time_alloc, rt.execution_time_copy_in,
                   rt.execution_time_launch, rt.execution_time_copy_out);
        }
    }

    if (test_results) {
        ret = compare_results(&gpu_results, &host_results, runtime_only);
    }

err:
    free(data_set);
    free(test_set);
    free(host_results.results);
    hipFree(gpu_results.results);
    hipFree(g_data_set);

    return (ret + gpu_ret + host_ret);
}
